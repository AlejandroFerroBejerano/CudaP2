#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define Filas 5
#define Columnas 7

__global__ void 
copiar(int *dev_a, int* dev_b)
{
  int position_x = blockIdx.x * blockDim.x + threadIdx.x;
  int position_y = blockDim.y * blockIdx.y + threadIdx.y;
  int position = position_y * Columnas + position_x;

  if (position_x >= Columnas || position_y >= Filas){
    /*pass*/
  }else{
    dev_b[position] = dev_a[position];
  }
}


void print_matriz(int resultado[Filas][Columnas]){
  int i,j;
    for (i=0; i< Filas; i++) {
	for(j=0; j< Columnas; j++){
	  printf("%d\t", resultado[i][j]);	
	}
     printf("\n");
  }
  printf("\n");
}

int
main(int argc, char** argv)
{
  int a[Filas][Columnas], b[Filas][Columnas];
  int *dev_a, *dev_b;
  int i,j,pos=0;
  dim3 nbloques(3,3);
  dim3 nhebras(3,2);

  hipMalloc((void**) &dev_a, Filas * Columnas * sizeof(int));
  hipMalloc((void**) &dev_b, Filas * Columnas * sizeof(int));
  
  // fill the arrays 'a' and 'b' on the CPU
  for (i=0; i<Filas; i++) {
	for(j=0; j<Columnas; j++){
		a[i][j]= pos++;
	}
  }

  hipMemcpy(dev_a, a, Filas * Columnas * sizeof(int), hipMemcpyHostToDevice);
  
  copiar<<<nbloques, nhebras>>>(dev_a, dev_b);

  hipMemcpy(b, dev_b, Filas * Columnas * sizeof(int), hipMemcpyDeviceToHost);
  
  printf("\nMatriz Origen\n");
  print_matriz(a);
  printf("\nMatriz Destino\n");
  print_matriz(b);

  hipFree(dev_a);
  hipFree(dev_b);

  return 0;
} 
