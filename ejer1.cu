#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define Filas 5
#define Columnas 7

__global__ void 
copiar(int *dev_a, int *dev_b)
{
  int position_x = blockIdx.x * blockDim.x + threadIdx.x + Columnas * threadIdx.y;
  int position_y = blockDim.y * blockIdx.y * Columnas;
	
  dev_b[position_x + position_y] = dev_a[position_x + position_y];
}

int
main(int argc, char** argv)
{
  int a[Filas][Columnas], b[Filas][Columnas];
  int *dev_a, *dev_b;
  int i,j,pos=0;
  dim3 nbloques(3,3);
  dim3 nhebras(3,2);

  hipMalloc((void**) &dev_a, Filas * Columnas * sizeof(int));
  hipMalloc((void**) &dev_b, Filas * Columnas * sizeof(int));
  
  // fill the arrays 'a' and 'b' on the CPU
  for (i=0; i<Filas; i++) {
	for(j=0; j<Columnas; j++){
		a[i][j]= pos++;
	}
  }

  hipMemcpy(dev_a, a, Filas * Columnas * sizeof(int), hipMemcpyHostToDevice);
  
  copiar<<<nbloques, nhebras>>>(dev_a, dev_b);

  hipMemcpy(b, dev_b, Filas * Columnas * sizeof(int), hipMemcpyDeviceToHost);
  pos = 0;
  for(i = 0; i < Filas; i++){
    for (j = 0; j< Columnas; j++){
    	printf("%d# %d -> %d \n",pos++, a[i][j], b[i][j]);
    }
  }

  hipFree(dev_a);
  hipFree(dev_b);

  return 0;
} 
