#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define Filas 5
#define Columnas 7
#define NbloquesX 3
#define NbloquesY 3
#define NhebrasX 3
#define NhebrasY 2

__global__ void 
trasponer(int *dev_a, int *dev_b)
{
  int position_x = blockIdx.x * blockDim.x + threadIdx.x;
  int position_y = blockDim.y * blockIdx.y + threadIdx.y;
  int position = position_y * Columnas + position_x;
  int position_reverse = position_x * Filas + position_y;

  if (position_x >= Columnas || position_y >= Filas){
    return;
  }else{
    dev_b[position_reverse] = dev_a[position];
  }
}


int
main(int argc, char** argv)
{
  int a[Filas][Columnas], b[Columnas][Filas];
  int *dev_a, *dev_b;
  int i,j,pos=0;
  dim3 nbloques(NbloquesX,NbloquesY);
  dim3 nhebras(NhebrasX,NhebrasY);

  hipMalloc((void**) &dev_a, Filas * Columnas * sizeof(int));
  hipMalloc((void**) &dev_b, Filas * Columnas * sizeof(int));
  
  // fill the arrays 'a' and 'b' on the CPU
  for (i=0; i<Filas; i++) {
	for(j=0; j<Columnas; j++){
		a[i][j]= pos++;
	}
  }

  hipMemcpy(dev_a, a, Filas * Columnas * sizeof(int), hipMemcpyHostToDevice);
  
  trasponer<<<nbloques, nhebras>>>(dev_a, dev_b);

  hipMemcpy(b, dev_b, Filas * Columnas * sizeof(int), hipMemcpyDeviceToHost);

  printf("\nMatriz Origen\n");
  for (i=0; i< Filas; i++) {
	for(j=0; j< Columnas; j++){
	  printf("%d\t", a[i][j]);	
	}
    printf("\n");
  }
  printf("\nMatriz Traspuesta\n");
  for (i=0; i< Columnas; i++) {
	for(j=0; j< Filas; j++){
	  printf("%d\t", b[i][j]);	
	}
    printf("\n");
  }

  hipFree(dev_a);
  hipFree(dev_b);

  return 0;
} 
