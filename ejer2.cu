#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define Filas 5
#define Columnas 7

__global__ void 
trasponer(int *dev_a, int *dev_b)
{
  int position_x = blockIdx.x * blockDim.x + threadIdx.x;
  int position_y = blockDim.y * blockIdx.y + threadIdx.y;
  int position = position_y * Columnas + position_x;

  if (position_x >= Columnas || position_y >= Filas){
    /*pass*/
  }else{
    dev_b[position] = dev_a[(Filas * Columnas -1)-position];
  }
}

int
main(int argc, char** argv)
{
  int a[Filas][Columnas], b[Filas][Columnas];
  int *dev_a, *dev_b;
  int i,j,pos=0;
  dim3 nbloques(3,3);
  dim3 nhebras(3,2);

  hipMalloc((void**) &dev_a, Filas * Columnas * sizeof(int));
  hipMalloc((void**) &dev_b, Filas * Columnas * sizeof(int));
  
  // fill the arrays 'a' and 'b' on the CPU
  for (i=0; i<Filas; i++) {
	for(j=0; j<Columnas; j++){
		a[i][j]= pos++;
	}
  }

  hipMemcpy(dev_a, a, Filas * Columnas * sizeof(int), hipMemcpyHostToDevice);
  
  trasponer<<<nbloques, nhebras>>>(dev_a, dev_b);

  hipMemcpy(b, dev_b, Filas * Columnas * sizeof(int), hipMemcpyDeviceToHost);
  pos=0;
  for(i = 0; i < Filas; i++){
    for (j = 0; j< Columnas; j++){
    	printf("%d# %d -> %d \n",pos++, a[i][j], b[i][j]);
    }
  }

  hipFree(dev_a);
  hipFree(dev_b);

  return 0;
} 
