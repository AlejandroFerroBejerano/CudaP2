#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define Filas 5
#define Columnas 7

__global__ void 
copiar(int *dev_a, int *dev_b)
{
	int blockx, blocky, position_x,position_y, position;
	int gridx = Columnas/blockDim.x + Columnas%blockDim.x;
  int gridy = Filas/blockDim.y + Filas%blockDim.y;
			
  for(blockx=0; blockx <= gridx; blockx++){
		for(blocky=0; blocky <= gridy; blocky++){
			position_x = blockx * blockDim.x + threadIdx.x;
  		position_y = blocky * blockDim.y + threadIdx.y;
			position = position_y * Columnas + position_x;
			if (position_x >= Columnas || position_y >= Filas){
    		continue;/*pass*/
  		}else{
    		dev_b[position] = dev_a[position];
  		}
		}
  }/*finfor*/
}

void print_matriz(int resultado[Filas][Columnas]){
  int i,j;
  for (i=0; i< Filas; i++) {
		for(j=0; j< Columnas; j++){
	 	 printf("%d\t", resultado[i][j]);	
		}
     printf("\n");
  }
  printf("\n");
}

int
main(int argc, char** argv)
{
  int a[Filas][Columnas], b[Filas][Columnas];
  int *dev_a, *dev_b;
  int i,j,pos=0;
  dim3 nbloques(1,1);
  dim3 nhebras(3,2);

  hipMalloc((void**) &dev_a, Filas * Columnas * sizeof(int));
  hipMalloc((void**) &dev_b, Filas * Columnas * sizeof(int));
  
  // fill the arrays 'a' and 'b' on the CPU
  for (i=0; i<Filas; i++) {
	for(j=0; j<Columnas; j++){
		a[i][j]= pos++;
	}
  }

  hipMemcpy(dev_a, a, Filas * Columnas * sizeof(int), hipMemcpyHostToDevice);
  
  copiar<<<nbloques, nhebras>>>(dev_a, dev_b);

  hipMemcpy(b, dev_b, Filas * Columnas * sizeof(int), hipMemcpyDeviceToHost);
 
  printf("\nMatriz Origen\n");
  print_matriz(a);
  printf("\nMatriz Destino\n");
  print_matriz(b);

  hipFree(dev_a);
  hipFree(dev_b);

  return 0;
} 
