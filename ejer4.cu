#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define Filas 5
#define Columnas 7

__global__ void 
copiar(int *dev_a, int *dev_b)
{
	int blockx, blocky, position_x,position_y, position,position_reverse;
	int gridx = Columnas/blockDim.x + Columnas%blockDim.x;
  int gridy = Filas/blockDim.y + Filas%blockDim.y;
			
  for(blockx=0; blockx <= gridx; blockx++){
		for(blocky=0; blocky <= gridy; blocky++){

			position_x = blockx * blockDim.x + threadIdx.x;
  		position_y = blocky * blockDim.y + threadIdx.y;
			position = position_y * Columnas + position_x;
			position_reverse = position_x * Filas + position_y;

			if (position_x >= Columnas || position_y >= Filas){
    		/*pass*/
  		}else{
    		dev_b[position_reverse] = dev_a[position];
  		}
		}
  }/*finfor*/
}

void print_matriz(int resultado[Filas][Columnas]){
  int i,j;
  for (i=0; i< Filas; i++) {
		for(j=0; j< Columnas; j++){
	 	 printf("%d\t", resultado[i][j]);	
		}
     printf("\n");
  }
  printf("\n");
}

int
main(int argc, char** argv)
{
  int a[Filas][Columnas], b[Columnas][Filas];
  int *dev_a, *dev_b;
  int i,j,pos=0;
  dim3 nbloques(1,1);
  dim3 nhebras(3,2);

  hipMalloc((void**) &dev_a, Filas * Columnas * sizeof(int));
  hipMalloc((void**) &dev_b, Filas * Columnas * sizeof(int));
  
  // fill the arrays 'a' and 'b' on the CPU
  for (i=0; i<Filas; i++) {
	for(j=0; j<Columnas; j++){
		a[i][j]= pos++;
	}
  }

  hipMemcpy(dev_a, a, Filas * Columnas * sizeof(int), hipMemcpyHostToDevice);
  
  copiar<<<nbloques, nhebras>>>(dev_a, dev_b);

  hipMemcpy(b, dev_b, Filas * Columnas * sizeof(int), hipMemcpyDeviceToHost);
 
  printf("\nMatriz Origen\n");
  for (i=0; i< Filas; i++) {
	for(j=0; j< Columnas; j++){
	  printf("%d\t", a[i][j]);	
	}
    printf("\n");
  }
  printf("\nMatriz Traspuesta\n");
  for (i=0; i< Columnas; i++) {
	for(j=0; j< Filas; j++){
	  printf("%d\t", b[i][j]);	
	}
    printf("\n");
  }

  hipFree(dev_a);
  hipFree(dev_b);

  return 0;
} 
