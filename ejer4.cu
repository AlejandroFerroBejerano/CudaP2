#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define Filas 5
#define Columnas 7

__global__ void 
copiar(int *dev_a, int *dev_b)
{
  int position_x = threadIdx.x;
  int position_y = threadIdx.y + blockDim.x * blockDim.y * threadIdx.y;
  int position = position_x + position_y;
  int desp = 0;
  int iter = (Columnas/blockDim.x + Columnas%blockDim.x) * (Filas/blockDim.y + Filas%blockDim.y);

  for(int i=0; i <= iter; i++){
    desp= i * blockDim.x;
    dev_b[position + desp] = dev_a[(Filas * Columnas -1) - (position + desp)];
  }
}

int
main(int argc, char** argv)
{
  int a[Filas][Columnas], b[Filas][Columnas];
  int *dev_a, *dev_b;
  int i,j,pos=0;
  dim3 nbloques(1,1);
  dim3 nhebras(3,2);

  hipMalloc((void**) &dev_a, Filas * Columnas * sizeof(int));
  hipMalloc((void**) &dev_b, Filas * Columnas * sizeof(int));
  
  // fill the arrays 'a' and 'b' on the CPU
  for (i=0; i<Filas; i++) {
	for(j=0; j<Columnas; j++){
		a[i][j]= pos++;
	}
  }

  hipMemcpy(dev_a, a, Filas * Columnas * sizeof(int), hipMemcpyHostToDevice);
  
  copiar<<<nbloques, nhebras>>>(dev_a, dev_b);

  hipMemcpy(b, dev_b, Filas * Columnas * sizeof(int), hipMemcpyDeviceToHost);
  pos = 0;
  for(i = 0; i < Filas; i++){
    for (j = 0; j< Columnas; j++){
    	printf("%d# %d -> %d \n",pos++, a[i][j], b[i][j]);
    }
  }

  hipFree(dev_a);
  hipFree(dev_b);

  return 0;
} 